#include "hip/hip_runtime.h"

#include"hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions 
#include"hip/device_functions.h"
#include "hip/hip_runtime.h"
#include"time.h"
#include<>
#include"hip/device_functions.h"
#include <sstream>



#define BLOCK_SIZE 16



__global__ void Add(float* A, float*B, float* C,int M,int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i<M&&j<N)
	{
		C[i + j*N] = A[i + j*N] + B[i + j*N];
	}

}

void add(float* A, float* B, float* C, int M, int N)
{
	float* A_b;
	float* B_b;
	float* C_b;

	hipMalloc((void**)&A_b, M * sizeof(float));
	hipMalloc((void**)&B_b, N * sizeof(float));
	hipMalloc((void**)&C_b, M*N * sizeof(float));

	hipMemcpy(A_b, A, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_b, B, M * sizeof(float), hipMemcpyHostToDevice);


	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((M + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);
	Add<< <dimGrid, dimBlock >> > (A_b, B_b, C_b, M, N);
	hipMemcpy(C, C_b, M*N * sizeof(float), hipMemcpyDeviceToHost);

}

